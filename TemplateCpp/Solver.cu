#include "hip/hip_runtime.h"
#pragma region Include The Self Header file
#include "Solver.h"
#include "Timer.h"
#include "Scoring.h"
#include "NeighborOperation.h"
#include "Parser.h"
#pragma endregion

#pragma region Include STL library
#include <algorithm>
#include <vector>
#include <execution>
#include <math.h>
#include <random>
#include <climits>
#include <chrono>
#include <mutex>
#include <atomic>
#include <ctime>
#pragma endregion

// Google OR tools comes here
#include "ortools/sat/cp_model.h"
#include "ortools/sat/model.h"
#include "ortools/sat/sat_parameters.pb.h"
#include "ortools/util/time_limit.h"

// OMP tools
#include <omp.h>
#include <iostream>

using namespace std;
using namespace operations_research;
using namespace operations_research::sat;

Solution solveByAdvancedGA(Dataset& dataset);

struct Possible3Para {
	int NumTeam2people;
	int NumTeam3people;
	int NumTeam4people;
};
vector<Possible3Para> possible3Paras;

void CreatePossible3Parameters(int T2, int T3, int T4, int limitPizza)
{
	CpModelBuilder cpModel;
	mutex mtx;
	const IntVar t2 = cpModel.NewIntVar(Domain(0, T2));
	const IntVar t3 = cpModel.NewIntVar(Domain(0, T3));
	const IntVar t4 = cpModel.NewIntVar(Domain(0, T4));

	// Means that t2 * 2 + t3 * 3 + t4 * 4 == NumPizzas
	cpModel.AddEquality(LinearExpr::ScalProd({ t2, t3, t4 }, { 2, 3, 4 }), limitPizza);
	if (limitPizza > 10) 
	{
		cpModel.AddLessOrEqual(t2, t3); // t2 <= t3
		cpModel.AddLessOrEqual(t3, t4); // t3 <= t4
	}
	
	Model model;  SatParameters parameter;
	parameter.set_max_time_in_seconds(10);
	//parameter.set_num_search_workers(8);
	parameter.set_enumerate_all_solutions(true);
	model.Add(NewSatParameters(parameter));
	
	// Create an atomic Boolean that will be periodically checked by the limit.
	std::atomic<bool> stopped(false);
	model.GetOrCreate<TimeLimit>()->RegisterExternalBooleanAsLimit(&stopped);
	int solutionLimit = 100;

	model.Add(NewFeasibleSolutionObserver([&](const CpSolverResponse& r) {
		Possible3Para newPossible;
		newPossible.NumTeam2people = SolutionIntegerValue(r, t2);
		newPossible.NumTeam3people = SolutionIntegerValue(r, t3);
		newPossible.NumTeam4people = SolutionIntegerValue(r, t4);
		lock_guard<mutex> lck(mtx);
		possible3Paras.push_back(newPossible);
		if (possible3Paras.size() >= solutionLimit) {
			stopped = true;
		}
	}));

	const CpSolverResponse respond = SolveCpModel(cpModel.Build(), &model);
}

Solution Solver::GetTheSolution(Dataset& dataset)
{
#pragma region Creating the possible list of possible number of assigned team
	possible3Paras.clear();
	if (dataset.Num2PeopleTeam * 2 + dataset.Num3PeopleTeam * 3
		+ dataset.Num4PeopleTeam * 4 <= dataset.NumPizzas) {
		Possible3Para only;
		only.NumTeam2people = dataset.Num2PeopleTeam;
		only.NumTeam3people = dataset.Num3PeopleTeam;
		only.NumTeam4people = dataset.Num4PeopleTeam;
		possible3Paras.push_back(only);
	}
	if (dataset.Num2PeopleTeam * 2 + dataset.Num3PeopleTeam * 3
		+ dataset.Num4PeopleTeam * 4 > dataset.NumPizzas)
	{
		CreatePossible3Parameters(dataset.Num2PeopleTeam, dataset.Num3PeopleTeam,
			dataset.Num4PeopleTeam, dataset.NumPizzas);
	}
#pragma endregion

#pragma region Solving by Genetic Algorithm
	Solution solution = solveByAdvancedGA(dataset);
	solution.chosenPizzas.shrink_to_fit();
#pragma endregion

	return solution;
}

Solution solveByAdvancedGA(Dataset& dataset)
{	
#pragma region Initialize Class And Setting
	mutex mtx;  cerr.sync_with_stdio(0);  cerr.tie(NULL);
	const int	POPULATION_SIZE = 200;
	const int	GENERATION = 10000;

#pragma region Initialize Class (Not touch)
	mt19937_64 rng;
	uniform_int_distribution<int> uid;
	uniform_real_distribution<double> urd(0.05, 0.1);
	uniform_real_distribution<double> changeRate(0.0, 1.0);

	Scoring scoring;  Parser parser;
	NeighborOperation neighborOperation;
	Timer timer;      int seconds;
#pragma endregion

#pragma endregion

#pragma region Declare a collection of populations And first solution score
	vector<Solution> population;
	Solution bestSolution;
	bestSolution.score = LLONG_MIN;
#pragma endregion

#pragma region Set the timer for the GA
	int length = dataset.NumPizzas;

	if (length <= 10)
	{
		seconds = 10;
	}
	else if (length <= 500)
	{
		seconds = 60;
	}
	else seconds = 420;
	seconds = 60;
#pragma endregion

#pragma region Read the previous best solution from previous run
	population = parser.ReadSolutionsFromPreviousRun(dataset.filename, dataset);
#pragma endregion
	
#pragma region Initialize the population for genetic algorithm
	uid = uniform_int_distribution<int>(0, possible3Paras.size() - 1);

	while (population.size() < POPULATION_SIZE) 
	{
		Solution solution = neighborOperation.RandomGenerate(dataset);
		random_device rd;  rng = mt19937_64(rd()); int id = uid(rng);
		solution.NumChosen2People = possible3Paras[id].NumTeam2people;
		solution.NumChosen3People = possible3Paras[id].NumTeam3people;
		solution.NumChosen4People = possible3Paras[id].NumTeam4people;
		population.push_back(solution);
	}
#pragma endregion

#pragma region Calculate the score of the initialize population
	for_each(execution::par, population.begin(), population.end(), [&dataset](Solution& solution) {
		Scoring scoring;
		Solution temp = solution;
		solution.score = scoring.CalculateScore(dataset, temp);
	});
#pragma endregion

#pragma region Initialize the timer and limit if cannot find better solution for a long period
	int NotFindBetterSolution = 0;
	const int LIMIT_TIME_NOT_FIND_BETTER_SOLUTION = GENERATION >> 1;
	timer.SetTheTimer(seconds);
#pragma endregion

	for (int generation = 0; generation < GENERATION; generation++)
	{
#pragma region update best solution, check the time if exceeded (not touch)
		int size = population.size();
		bool findBetterSolution = false;

		for (int i = 0; i < size; i++) {
			if (population[i].score > bestSolution.score) 
			{
				bestSolution = population[i];
				findBetterSolution = true;
				NotFindBetterSolution = 0;
			}
		}
		if (!findBetterSolution) {
			NotFindBetterSolution++;
			if (NotFindBetterSolution > LIMIT_TIME_NOT_FIND_BETTER_SOLUTION) { break; }
		}
		if (timer.CheckTimerFinish()) { break; }
		if (NotFindBetterSolution > 50) { break; }
#pragma endregion

#pragma region Prepare for Wheel Selection
		long long sum = 0;
		vector<double> wheelP;

		for (int i = 0; i < size; i++) 
		{
			sum = sum + population[i].score;
			wheelP.push_back(0);
		}
		for (int i = 0; i < size; i++) 
		{
			double p = (double)population[i].score / sum;
			wheelP[i] = p;
		}
		for (int i = 1; i < size; i++) wheelP[i] += wheelP[i - 1];
#pragma endregion

#pragma region Add 10% best from parent to new generation
		vector<Solution> newGeneration;       newGeneration.clear();
		//newGeneration.push_back(bestSolution);

		//int s = POPULATION_SIZE / 10;  // Choose 10% elite parents
		//for (int i = 0; i < s; i++)
		//{
		//	newGeneration.push_back(population[i]);
		//}
#pragma endregion

#pragma region Crossover some parents of current population, create new generation population
		int s = POPULATION_SIZE * 0.4;    // 80% will be choose based on the offspring
		uid = uniform_int_distribution<int>(0, population.size() >> 1);
		uniform_real_distribution<double> uwheel(0.0, wheelP[wheelP.size() - 1]);

		#pragma omp parallel for
		for (int i = 0; i < s; i++)
		{
			random_device rd;   rng = mt19937_64(rd());
			int p1 = lower_bound(wheelP.begin(), wheelP.end(), uwheel(rng)) - wheelP.begin();
			int p2 = lower_bound(wheelP.begin(), wheelP.end(), uwheel(rng)) - wheelP.begin();

			Solution s1 = population[p1];
			Solution s2 = population[p2];
			neighborOperation.CrossOver(s1, s2);

			lock_guard<mutex> lck(mtx);
			newGeneration.push_back(s1);
			newGeneration.push_back(s2);
		}
#pragma endregion

#pragma region Mutation some parts of new population
		random_device rd;
		rng = mt19937_64(rd()); // 0.5% to 1% will mutate offspring
		double percentage = urd(rng);
		int currentNewPopulation = newGeneration.size();
		int MumberOfMutation = currentNewPopulation * percentage;
		uid = uniform_int_distribution<int>(0, currentNewPopulation - 1);
		cerr << NotFindBetterSolution << "\n";

		//#pragma omp parallel for
		for (int i = 0; i < MumberOfMutation; i++)
		{
			random_device rd;
			rng = mt19937_64(rd());  
			int id = uid(rng);

			Solution temporary = newGeneration[id];

			if (NotFindBetterSolution >= 5)
			{
				int limit = min(NotFindBetterSolution / 5, 10);
				for (int j = 0; j < limit; j++)
				{
					shuffle(temporary.chosenPizzas.begin(),
							temporary.chosenPizzas.end(), rng);
				}
			}
			else
			{
				temporary = neighborOperation.Mutate(temporary);
			}

			double rate = changeRate(rng);
			if (rate >= 0.5)
			{
				uniform_int_distribution<int> f(0, possible3Paras.size() - 1);
				random_device rd;  rng = mt19937_64(rd());  int id = f(rng);
				temporary.NumChosen2People = possible3Paras[id].NumTeam2people;
				temporary.NumChosen3People = possible3Paras[id].NumTeam3people;
				temporary.NumChosen4People = possible3Paras[id].NumTeam4people;
			}

			//lock_guard<mutex> lck(mtx);
			newGeneration[id] = temporary;
		}
#pragma endregion
		if (NotFindBetterSolution > 10) {
			for (int i = 0; i < NotFindBetterSolution / 10; i++) {
				random_device rd;
				rng = mt19937_64(rd());
				shuffle(newGeneration[i].chosenPizzas.begin(),
					newGeneration[i].chosenPizzas.end(), rng);
			}
		}
#pragma region Generating random population for diversity
		s = POPULATION_SIZE / 10;    // 10% from Random solution
		uid = uniform_int_distribution<int>(0, possible3Paras.size() - 1);

		#pragma omp parallel for
		for (int i = 0; i < s; i++)
		{
			random_device rd;
			rng = mt19937_64(rd());   int id = uid(rng);
			Solution newOffspring = neighborOperation.RandomGenerate(dataset);

			newOffspring.NumChosen2People = possible3Paras[id].NumTeam2people;
			newOffspring.NumChosen3People = possible3Paras[id].NumTeam3people;
			newOffspring.NumChosen4People = possible3Paras[id].NumTeam4people;

			lock_guard<mutex> lck(mtx);
			newGeneration.push_back(newOffspring);
		}
#pragma endregion

#pragma region Get the score of new generation population
		for_each(execution::par, newGeneration.begin(), newGeneration.end(), [&dataset](Solution& solution)
			{
				Scoring scoring;         Solution temp = solution;
				solution.score = scoring.CalculateScore(dataset, temp);
			});
#pragma endregion

#pragma region Write the Debug info
		cerr << generation << " " << population[0].score << " " << bestSolution.score << "\n";
#pragma endregion

		population = newGeneration;
	}
#pragma region Return the best solution that have find so far (not touch)
	for (int i = 0; i < population.size(); i++) {
		if (population[i].score > bestSolution.score)
		{
			bestSolution = population[i];
		}
	}
	return bestSolution;
#pragma endregion
}

/// <summary>
/// This is the main implementation for the Google Hash Code Problem
/// Calculating from the dataset, and transfer into solution using all 
/// </summary>

// Skip this part below
Solver::Solver()
{
}
